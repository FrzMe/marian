#include "enc_out_gpu.h"
#include "gpu/mblas/matrix_functions.h"

using namespace std;

namespace amunmt {
namespace GPU {
namespace mblas {

EncOutGPU::EncOutGPU(SentencesPtr sentences)
:EncOut(sentences)
{
  size_t tab = 0;
  size_t maxSentenceLength = sentences->GetMaxLength();

  //cerr << "1dMapping=" << mblas::Debug(dMapping, 2) << endl;
  HostVector<uint> hSentenceLengths(sentences->size());

  for (size_t i = 0; i < sentences->size(); ++i) {
    const Sentence &sentence = *sentences->at(i);
    hSentenceLengths[i] = sentence.GetWords(tab).size();
  }

  sentenceLengths_.NewSize(sentences->size(), 1, 1, 1);
  mblas::copy(thrust::raw_pointer_cast(hSentenceLengths.data()),
              hSentenceLengths.size(),
              sentenceLengths_.data(),
              hipMemcpyHostToDevice);

  //cerr << "sentenceLengths_=" << sentenceLengths_.Debug(2) << endl;
  //cerr << "sentencesMask_=" << sentencesMask_.Debug(2) << endl;
}

}
}
}

