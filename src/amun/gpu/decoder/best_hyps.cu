#include "best_hyps.h"

namespace amunmt {
namespace GPU {

void BestHyps::CalcBeam
(
    const Beam& prevHyps,
    Scorer& scorer,
    const Words& filterIndices,
    std::vector<Beam>& beams,
    std::vector<uint>& beamSizes)
{
  BEGIN_TIMER("CalcBeam");

  using namespace mblas;

  mblas::Matrix& Probs = static_cast<mblas::Matrix&>(scorer.GetProbs());

  HostVector<float> vCosts;
  for (auto& h : prevHyps) {
    vCosts.push_back(h->GetCost());
  }
  mblas::copy(vCosts.begin(), vCosts.end(), Costs.begin());

  const bool isFirst = (vCosts[0] == 0.0f) ? true : false;

  BroadcastVecColumn(_1 + _2, Probs, Costs);

  if (forbidUNK_) {
    DisAllowUNK(Probs);
  }

  size_t beamSizeSum = std::accumulate(beamSizes.begin(), beamSizes.end(), 0);

  std::vector<float> bestCosts;
  std::vector<unsigned> bestKeys;

  FindBests(beamSizes, Probs, bestCosts, bestKeys, isFirst);

  std::vector<HostVector<float>> breakDowns;
  if (returnNBestList_) {
      breakDowns.push_back(bestCosts);
  }

  std::map<size_t, size_t> batchMap;
  size_t tmp = 0;
  for (size_t batchID = 0; batchID < beamSizes.size(); ++batchID) {
    for (size_t t = 0; t < beamSizes[batchID]; ++t) {
      batchMap[tmp++] = batchID;
    }
  }

  for (size_t i = 0; i < beamSizeSum; i++) {
    size_t wordIndex = bestKeys[i] % Probs.dim(1);
    if (isInputFiltered_) {
      wordIndex = filterIndices[wordIndex];
    }

    size_t hypIndex  = bestKeys[i] / Probs.dim(1);
    float cost = bestCosts[i];

    HypothesisPtr prevHyp = prevHyps.at(hypIndex);
    HypothesisPtr hyp;
    if (returnAttentionWeights_) {
      hyp.reset(new Hypothesis(prevHyp, wordIndex, hypIndex, cost,
                               GetAlignments(scorer, hypIndex)));
    } else {
      hyp.reset(new Hypothesis(prevHyp, wordIndex, hypIndex, cost));
    }

    if(returnNBestList_) {
      hyp->GetCostBreakdown().resize(1);
      float sum = 0;

      hyp->GetCostBreakdown()[0] = breakDowns[0][i];

      hyp->GetCostBreakdown()[0] -= sum;
    }

    beams[batchMap[i]].push_back(hyp);

  }

  PAUSE_TIMER("CalcBeam");

}

std::vector<SoftAlignmentPtr> BestHyps::GetAlignments(Scorer& scorer, size_t hypIndex)
{
  std::vector<SoftAlignmentPtr> alignments;

  if (GPU::EncoderDecoder* encdec = dynamic_cast<GPU::EncoderDecoder*>(&scorer)) {
    const mblas::Matrix &attention = encdec->GetAttention();
    size_t attLength = attention.dim(1);

    SoftAlignment *softAlignment = new SoftAlignment(attLength);
    mblas::copy(
        attention.data() + hypIndex * attLength,
        attLength,
        thrust::raw_pointer_cast(softAlignment->data()),
        hipMemcpyDeviceToHost
    );

    alignments.emplace_back(softAlignment);

  } else {
    amunmt_UTIL_THROW2("Return Alignment is allowed only with Nematus scorer.");
  }

  return alignments;
}

}
}


