#include "beam_size_gpu.h"
#include "gpu/mblas/matrix_functions.h"

using namespace std;

namespace amunmt {
namespace GPU {

BeamSizeGPU::BeamSizeGPU(EncParamsPtr encParams)
:BeamSize(encParams->sentences)
,sentencesMask(encParams->GetSentenceMask2<mblas::IMatrix>())
,sourceContext(encParams->GetSourceContext2<mblas::Matrix>())
{

}

void BeamSizeGPU::Init(EncParamsPtr encParams)
{
  BeamSize::Init(encParams->sentences);
  //sentencesMask = encParams->GetSentenceMask2<mblas::IMatrix>());
  //sourceContext = encParams->GetSourceContext2<mblas::Matrix>());
}

void BeamSizeGPU::DeleteEmpty()
{
  size_t i = 0;
  while (i < size()) {
    if (sizes_[i]) {
      ++i;
    }
    else {
      sizes_.erase(sizes_.begin() + i);
      sentences_.erase(sentences_.begin() + i);

      cerr << "DELETE " << i;

      HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
      cerr << " sentencesMask=" << sentencesMask.Debug(0) << flush;
      Delete1Axis(sentencesMask, 1, i);
      HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
      cerr << " " << sentencesMask.Debug(0) << flush;

      HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
      cerr << " sourceContext=" << sourceContext.Debug(0) << flush;
      Delete1Axis(sourceContext, 3, i);
      HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
      cerr << " " << sourceContext.Debug(0) << flush;

    }
  }
}

std::string BeamSizeGPU::Debug(size_t verbosity) const
{
  stringstream strm;

  strm << amunmt::BeamSize::Debug(verbosity);
  strm << " sentencesMask=" << sentencesMask.Debug(0);
  strm << " sourceContext=" << sourceContext.Debug(0);

  return strm.str();
}

}
}

