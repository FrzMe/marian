// -*- mode: c++; tab-width: 2; indent-tabs-mode: nil -*-
#include <iostream>

#include "common/god.h"
#include "common/sentences.h"

#include "encoder_decoder.h"
#include "gpu/mblas/matrix_functions.h"
#include "gpu/dl4mt/dl4mt.h"
#include "gpu/decoder/encoder_decoder_state.h"
#include "gpu/decoder/best_hyps.h"

using namespace std;

namespace amunmt {
namespace GPU {

void ttt()
{
  cerr << "ttt" << endl;
}

void EncoderDecoder::Decode()
{
  cerr << "Decode" << endl;

  //mblas::EncParamsPtr encParams = encDecBuffer_.remove();
  //cerr << "BeginSentenceState encParams->sourceContext_=" << encParams->sourceContext_.Debug(0) << endl;

}

///////////////////////////////////////////////////////////////////////////////
EncoderDecoder::EncoderDecoder(
		const God &god,
		const std::string& name,
        const YAML::Node& config,
        size_t tab,
        const Weights& model)
  : Scorer(god, name, config, tab),
    model_(model),
    encoder_(new Encoder(model_)),
    decoder_(new Decoder(god, model_)),
    indices_(god.Get<size_t>("beam-size"))
{
  std::thread *thread = new std::thread( [&]{ Decode(); });
  decThread_.reset(thread);

}

EncoderDecoder::~EncoderDecoder()
{
  decThread_->join();
}

State* EncoderDecoder::NewState() const {
  return new EDState();
}

void EncoderDecoder::Encode(const SentencesPtr source) {
  BEGIN_TIMER("SetSource");

  mblas::EncParamsPtr encParams(new mblas::EncParams());
  encParams->sentences = source;

  encoder_->Encode(*source, tab_, encParams);

  encDecBuffer_.add(encParams);
  cerr << "Encode encParams->sourceContext_=" << encParams->sourceContext_.Debug(0) << endl;

  PAUSE_TIMER("SetSource");
}

void EncoderDecoder::BeginSentenceState(State& state, size_t batchSize)
{
  mblas::EncParamsPtr encParams = encDecBuffer_.remove();
  cerr << "BeginSentenceState encParams->sourceContext_=" << encParams->sourceContext_.Debug(0) << endl;
  cerr << "BeginSentenceState encParams->sentencesMask_=" << encParams->sentencesMask_.Debug(0) << endl;
  cerr << "batchSize=" << batchSize << endl;

  EDState& edState = state.get<EDState>();

  decoder_->EmptyState(edState.GetStates(), encParams, batchSize);

  decoder_->EmptyEmbedding(edState.GetEmbeddings(), batchSize);
}

void EncoderDecoder::Decode(const State& in, State& out, const std::vector<uint>& beamSizes) {
  BEGIN_TIMER("Decode");
  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();

  decoder_->Decode(edOut.GetStates(),
                     edIn.GetStates(),
                     edIn.GetEmbeddings(),
                     beamSizes);
  PAUSE_TIMER("Decode");
}

void EncoderDecoder::AssembleBeamState(const State& in,
                               const Beam& beam,
                               State& out) {
  std::vector<size_t> beamWords;
  std::vector<uint> beamStateIds;
  for (const HypothesisPtr &h : beam) {
     beamWords.push_back(h->GetWord());
     beamStateIds.push_back(h->GetPrevStateIndex());
  }
  //cerr << "beamWords=" << Debug(beamWords, 2) << endl;
  //cerr << "beamStateIds=" << Debug(beamStateIds, 2) << endl;

  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();
  indices_.resize(beamStateIds.size());
  HostVector<uint> tmp = beamStateIds;

  mblas::copy(thrust::raw_pointer_cast(tmp.data()),
      beamStateIds.size(),
      thrust::raw_pointer_cast(indices_.data()),
      hipMemcpyHostToDevice);
  //cerr << "indices_=" << mblas::Debug(indices_, 2) << endl;

  mblas::Assemble(edOut.GetStates(), edIn.GetStates(), indices_);
  //cerr << "edOut.GetStates()=" << edOut.GetStates().Debug(1) << endl;

  //cerr << "beamWords=" << Debug(beamWords, 2) << endl;
  decoder_->Lookup(edOut.GetEmbeddings(), beamWords);
  //cerr << "edOut.GetEmbeddings()=" << edOut.GetEmbeddings().Debug(1) << endl;
}

void EncoderDecoder::GetAttention(mblas::Matrix& Attention) {
  decoder_->GetAttention(Attention);
}

BaseMatrix& EncoderDecoder::GetProbs() {
  return decoder_->GetProbs();
}

mblas::Matrix& EncoderDecoder::GetAttention() {
  return decoder_->GetAttention();
}

size_t EncoderDecoder::GetVocabSize() const {
  return decoder_->GetVocabSize();
}

void EncoderDecoder::Filter(const std::vector<size_t>& filterIds) {
  decoder_->Filter(filterIds);
}


}
}

