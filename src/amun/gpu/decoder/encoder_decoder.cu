// -*- mode: c++; tab-width: 2; indent-tabs-mode: nil -*-
#include <iostream>

#include "common/god.h"
#include "common/sentences.h"
#include "common/search.h"

#include "encoder_decoder.h"
#include "gpu/mblas/matrix_functions.h"
#include "gpu/dl4mt/dl4mt.h"
#include "gpu/decoder/encoder_decoder_state.h"
#include "gpu/decoder/best_hyps.h"

using namespace std;

namespace amunmt {
namespace GPU {

void EncoderDecoder::DecodeAsync(const God &god)
{
  return;

  OutputCollector &outputCollector = god.GetOutputCollector();

  while (true) {
    mblas::EncParamsPtr encParams = encDecBuffer_.remove();
    assert(encParams.get());
    assert(encParams->sentences.get());
    cerr << "BeginSentenceState encParams->sourceContext_=" << encParams->sourceContext_.Debug(0) << endl;

    HistoriesPtr histories = DecodeAsync(encParams);

    for (size_t i = 0; i < histories->size(); ++i) {
      const History &history = *histories->at(i);
      size_t lineNum = history.GetLineNum();

      std::stringstream strm;
      search_.Printer(god, history, strm);

      outputCollector.Write(lineNum, strm.str());
    }


  }
}

HistoriesPtr EncoderDecoder::DecodeAsync(mblas::EncParamsPtr encParams)
{
  boost::timer::cpu_timer timer;
  cerr << "DecodeAsync" << endl;

  // begin decoding - create 1st decode states
  State *state = NewState();
  BeginSentenceState(*state, encParams->sentences->size());

  State *nextState = NewState();
  std::vector<uint> beamSizes(encParams->sentences->size(), 1);

  HistoriesPtr histories(new Histories(*encParams->sentences, search_.NormalizeScore()));
  Beam prevHyps = histories->GetFirstHyps();

  for (size_t decoderStep = 0; decoderStep < 3 * encParams->sentences->GetMaxLength(); ++decoderStep) {
    // decode
    Decode(*state, *nextState, beamSizes);

    // beams
    if (decoderStep == 0) {
      for (auto& beamSize : beamSizes) {
        beamSize = search_.MaxBeamSize();
      }
    }

    size_t batchSize = beamSizes.size();
    Beams beams(batchSize);
    search_.BestHyps()->CalcBeam(prevHyps, *this, search_.FilterIndices(), beams, beamSizes);
    histories->Add(beams);

    Beam survivors;
    for (size_t batchId = 0; batchId < batchSize; ++batchId) {
      for (auto& h : beams[batchId]) {
        if (h->GetWord() != EOS_ID) {
          survivors.push_back(h);
        } else {
          --beamSizes[batchId];
        }
      }
    }

    if (survivors.size() == 0) {
      return histories;
    }

    AssembleBeamState(*nextState, survivors, *state);

    prevHyps.swap(survivors);

  }

  CleanUpAfterSentence();

  LOG(progress)->info("Search took {}", timer.format(3, "%ws"));
  return histories;
}

///////////////////////////////////////////////////////////////////////////////
EncoderDecoder::EncoderDecoder(
        const God &god,
        const std::string& name,
        const YAML::Node& config,
        size_t tab,
        const Weights& model,
        const Search &search)
  : Scorer(god, name, config, tab, search),
    model_(model),
    encoder_(new Encoder(model_)),
    decoder_(new Decoder(god, model_)),
    indices_(god.Get<size_t>("beam-size"))
{
  std::thread *thread = new std::thread( [&]{ DecodeAsync(god); });
  decThread_.reset(thread);

}

EncoderDecoder::~EncoderDecoder()
{
  decThread_->join();
}

State* EncoderDecoder::NewState() const {
  return new EDState();
}

void EncoderDecoder::Encode(const SentencesPtr source) {
  BEGIN_TIMER("SetSource");

  mblas::EncParamsPtr encParams(new mblas::EncParams());
  encParams->sentences = source;

  encoder_->Encode(*source, tab_, encParams);

  encDecBuffer_.add(encParams);
  cerr << "Encode encParams->sourceContext_=" << encParams->sourceContext_.Debug(0) << endl;

  PAUSE_TIMER("SetSource");
}

void EncoderDecoder::BeginSentenceState(State& state, size_t batchSize)
{
  mblas::EncParamsPtr encParams = encDecBuffer_.remove();
  cerr << "BeginSentenceState encParams->sourceContext_=" << encParams->sourceContext_.Debug(0) << endl;
  cerr << "BeginSentenceState encParams->sentencesMask_=" << encParams->sentencesMask_.Debug(0) << endl;
  cerr << "batchSize=" << batchSize << endl;

  EDState& edState = state.get<EDState>();

  decoder_->EmptyState(edState.GetStates(), encParams, batchSize);

  decoder_->EmptyEmbedding(edState.GetEmbeddings(), batchSize);
}

void EncoderDecoder::Decode(const State& in, State& out, const std::vector<uint>& beamSizes) {
  BEGIN_TIMER("Decode");
  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();

  decoder_->Decode(edOut.GetStates(),
                     edIn.GetStates(),
                     edIn.GetEmbeddings(),
                     beamSizes);
  PAUSE_TIMER("Decode");
}

void EncoderDecoder::AssembleBeamState(const State& in,
                               const Beam& beam,
                               State& out) {
  std::vector<size_t> beamWords;
  std::vector<uint> beamStateIds;
  for (const HypothesisPtr &h : beam) {
     beamWords.push_back(h->GetWord());
     beamStateIds.push_back(h->GetPrevStateIndex());
  }
  //cerr << "beamWords=" << Debug(beamWords, 2) << endl;
  //cerr << "beamStateIds=" << Debug(beamStateIds, 2) << endl;

  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();
  indices_.resize(beamStateIds.size());
  HostVector<uint> tmp = beamStateIds;

  mblas::copy(thrust::raw_pointer_cast(tmp.data()),
      beamStateIds.size(),
      thrust::raw_pointer_cast(indices_.data()),
      hipMemcpyHostToDevice);
  //cerr << "indices_=" << mblas::Debug(indices_, 2) << endl;

  mblas::Assemble(edOut.GetStates(), edIn.GetStates(), indices_);
  //cerr << "edOut.GetStates()=" << edOut.GetStates().Debug(1) << endl;

  //cerr << "beamWords=" << Debug(beamWords, 2) << endl;
  decoder_->Lookup(edOut.GetEmbeddings(), beamWords);
  //cerr << "edOut.GetEmbeddings()=" << edOut.GetEmbeddings().Debug(1) << endl;
}

void EncoderDecoder::GetAttention(mblas::Matrix& Attention) {
  decoder_->GetAttention(Attention);
}

BaseMatrix& EncoderDecoder::GetProbs() {
  return decoder_->GetProbs();
}

mblas::Matrix& EncoderDecoder::GetAttention() {
  return decoder_->GetAttention();
}

size_t EncoderDecoder::GetVocabSize() const {
  return decoder_->GetVocabSize();
}

void EncoderDecoder::Filter(const std::vector<size_t>& filterIds) {
  decoder_->Filter(filterIds);
}


}
}

