// -*- mode: c++; tab-width: 2; indent-tabs-mode: nil -*-
#include <iostream>

#include "common/god.h"
#include "common/sentences.h"
#include "common/search.h"
#include "common/histories.h"

#include "encoder_decoder.h"
#include "gpu/mblas/matrix_functions.h"
#include "gpu/dl4mt/dl4mt.h"
#include "gpu/decoder/encoder_decoder_state.h"
#include "gpu/decoder/best_hyps.h"
#include "gpu/decoder/beam_size_gpu.h"

using namespace std;

namespace amunmt {
namespace GPU {

///////////////////////////////////////////////////////////////////////////////
EncoderDecoder::EncoderDecoder(
        const God &god,
        const std::string& name,
        const YAML::Node& config,
        size_t tab,
        const Weights& model,
        const Search &search)
: Scorer(god, name, config, tab, search),
  model_(model),
  encoder_(new Encoder(model_)),
  decoder_(new Decoder(god, model_)),
  indices_(god.Get<size_t>("beam-size")),
  encDecBuffer_(3)

{
  std::thread *thread = new std::thread( [&]{ DecodeAsync(god); });
  decThread_.reset(thread);

}

EncoderDecoder::~EncoderDecoder()
{
  decThread_->join();
}

State* EncoderDecoder::NewState() const {
  return new EDState();
}

void EncoderDecoder::Encode(const SentencesPtr source) {
  BEGIN_TIMER("Encode");

  mblas::EncParamsPtr encParams(new mblas::EncParamsGPU());
  encParams->sentences = source;

  if (source->size()) {
    encoder_->Encode(*source, tab_, encParams);
  }

  encDecBuffer_.add(encParams);
  //cerr << "Encode encParams->sourceContext_=" << encParams->sourceContext_.Debug(0) << endl;

  PAUSE_TIMER("Encode");
}

void EncoderDecoder::BeginSentenceState(State& state, size_t batchSize, mblas::EncParamsPtr encParams)
{
  //cerr << "BeginSentenceState encParams->sourceContext_=" << encParams->sourceContext_.Debug(0) << endl;
  //cerr << "BeginSentenceState encParams->sentencesMask_=" << encParams->sentencesMask_.Debug(0) << endl;
  //cerr << "batchSize=" << batchSize << endl;

  EDState& edState = state.get<EDState>();

  decoder_->EmptyState(edState.GetStates(), encParams, batchSize);

  decoder_->EmptyEmbedding(edState.GetEmbeddings(), batchSize);
}

void EncoderDecoder::Decode(const State& in, State& out, const BeamSize& beamSizes) {
  BEGIN_TIMER("Decode");
  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();

  const BeamSizeGPU &bs = static_cast<const BeamSizeGPU&>(beamSizes);

  decoder_->Decode(edOut.GetStates(),
                     edIn.GetStates(),
                     edIn.GetEmbeddings(),
                     bs);
  PAUSE_TIMER("Decode");
}


void EncoderDecoder::DecodeAsync(const God &god)
{
  //cerr << "BeginSentenceState encParams->sourceContext_=" << encParams->sourceContext_.Debug(0) << endl;
  try {
    DecodeAsyncInternal(god);
  }
  catch(thrust::system_error &e)
  {
    std::cerr << "CUDA error during some_function: " << e.what() << std::endl;
    abort();
  }
  catch(std::bad_alloc &e)
  {
    std::cerr << "Bad memory allocation during some_function: " << e.what() << std::endl;
    abort();
  }
  catch(std::runtime_error &e)
  {
    std::cerr << "Runtime error during some_function: " << e.what() << std::endl;
    abort();
  }
  catch(...)
  {
    std::cerr << "Some other kind of error during some_function" << std::endl;
    abort();
  }
}

void EncoderDecoder::DecodeAsyncInternal(const God &god)
{
  while (true) {
    mblas::EncParamsPtr encParams = encDecBuffer_.remove();
    assert(encParams.get());
    assert(encParams->sentences.get());

    if (encParams->sentences->size() == 0) {
      return;
    }

    boost::timer::cpu_timer timer;

    // begin decoding - create 1st decode states
    State *state = NewState();
    BeginSentenceState(*state, encParams->sentences->size(), encParams);

    State *nextState = NewState();

    Histories histories(new BeamSizeGPU(encParams), search_.NormalizeScore());
    Hypotheses prevHyps = histories.GetFirstHyps();

    cerr << "beamSizes1=" << histories.GetBeamSizes().Debug(2) << endl;

    // decode
    for (size_t decoderStep = 0; decoderStep < 3 * encParams->sentences->GetMaxLength(); ++decoderStep) {
      boost::timer::cpu_timer timerStep;

      //cerr << "beamSizes2=" << beamSizes.Debug(2) << endl;
      Decode(*state, *nextState, histories.GetBeamSizes());

      cerr << "beamSizes3=" << histories.GetBeamSizes().Debug(2) << endl;
      cerr << "state=" << state->Debug(0) << endl;

      // beams
      if (decoderStep == 0) {
        histories.InitBeamSize(search_.MaxBeamSize());
      }
      //cerr << "beamSizes4=" << beamSizes.Debug(2) << endl;

      Beams beams;
      search_.BestHyps()->CalcBeam(prevHyps, *this, search_.FilterIndices(), beams, histories.GetBeamSizes());

      Hypotheses survivors = histories.AddAndOutput(god, beams);

      cerr << "beamSizes5=" << histories.GetBeamSizes().Debug(2) << endl;

      /*
      cerr << "beamSizes=" << Debug(beamSizes, 2) << endl;
      cerr << "survivors=" << survivors.size() << endl;
      cerr << "beams=" << beams.size() << endl;
      cerr << "histories=" << histories.size() << endl;
      cerr << "state=" << state->Debug(0) << endl;
      cerr << "nextState=" << nextState->Debug(0) << endl;
      */

      if (survivors.size() == 0) {
        break;
      }

      AssembleBeamState(*nextState, survivors, *state);

      //beamSizes.DeleteEmpty();
      //cerr << "beamSizes6=" << beamSizes.Debug(2) << endl;

      prevHyps.swap(survivors);

      cerr << endl;
      LOG(progress)->info("Step took {}", timerStep.format(3, "%ws"));
    } // for (size_t decoderStep = 0; decoderStep < 3 * encParams->sentences->GetMaxLength(); ++decoderStep) {

    histories.OutputRemaining(god);

    CleanUpAfterSentence();

    // output
    //Output(god, histories);

    LOG(progress)->info("Decoding took {}", timer.format(3, "%ws"));
  }
}


void EncoderDecoder::AssembleBeamState(const State& in,
                               const Hypotheses& hypos,
                               State& out) {
  std::vector<size_t> beamWords;
  std::vector<uint> beamStateIds;
  for (const HypothesisPtr &h : hypos) {
     beamWords.push_back(h->GetWord());
     beamStateIds.push_back(h->GetPrevStateIndex());
  }
  //cerr << "beamWords=" << Debug(beamWords, 2) << endl;
  //cerr << "beamStateIds=" << Debug(beamStateIds, 2) << endl;

  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();
  indices_.resize(beamStateIds.size());
  HostVector<uint> tmp = beamStateIds;

  mblas::copy(thrust::raw_pointer_cast(tmp.data()),
      beamStateIds.size(),
      thrust::raw_pointer_cast(indices_.data()),
      hipMemcpyHostToDevice);
  //cerr << "indices_=" << mblas::Debug(indices_, 2) << endl;

  mblas::Assemble(edOut.GetStates(), edIn.GetStates(), indices_);
  //cerr << "edOut.GetStates()=" << edOut.GetStates().Debug(1) << endl;

  //cerr << "beamWords=" << Debug(beamWords, 2) << endl;
  decoder_->Lookup(edOut.GetEmbeddings(), beamWords);
  //cerr << "edOut.GetEmbeddings()=" << edOut.GetEmbeddings().Debug(1) << endl;
}

void EncoderDecoder::GetAttention(mblas::Matrix& Attention) {
  decoder_->GetAttention(Attention);
}

BaseMatrix& EncoderDecoder::GetProbs() {
  return decoder_->GetProbs();
}

mblas::Matrix& EncoderDecoder::GetAttention() {
  return decoder_->GetAttention();
}

size_t EncoderDecoder::GetVocabSize() const {
  return decoder_->GetVocabSize();
}

void EncoderDecoder::Filter(const std::vector<size_t>& filterIds) {
  decoder_->Filter(filterIds);
}


}
}

