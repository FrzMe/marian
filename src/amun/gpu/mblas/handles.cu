#include "handles.h"
#include "gpu/types-gpu.h"

namespace amunmt {
namespace GPU {
namespace mblas {

CudaStreamHandler::CudaStreamHandler()
{
  HANDLE_ERROR( hipStreamCreate(&stream_));
  HANDLE_ERROR( hipStreamCreate(&streamEnc_));
  // hipStreamCreateWithFlags(stream_.get(), hipStreamNonBlocking);
}

CudaStreamHandler::~CudaStreamHandler()
{
  HANDLE_ERROR(hipStreamDestroy(stream_));
  HANDLE_ERROR(hipStreamDestroy(streamEnc_));
}

///////////////////////////////////////////////////////////////////////
CublasHandler::CublasHandler()
{
  CreateHandle(handle_, CudaStreamHandler::GetStream());
  CreateHandle(handleEnc_, CudaStreamHandler::GetEncoderStream());
}

CublasHandler::~CublasHandler() {
  hipblasDestroy(handle_);
}

void CublasHandler::CreateHandle(hipblasHandle_t &handle, const hipStream_t &stream) const
{
  hipblasStatus_t stat;
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
  printf ("hipblasCreate initialization failed\n");
  abort();
  }

  stat = hipblasSetStream(handle, stream);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
  printf ("hipblasSetStream initialization failed\n");
  abort();
  }

}


}
}
}
