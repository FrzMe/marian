#include "handles.h"
#include "gpu/types-gpu.h"

namespace amunmt {
namespace GPU {
namespace mblas {

CudaStreamHandler::CudaStreamHandler()
{
  HANDLE_ERROR( hipStreamCreate(&stream_));
  HANDLE_ERROR( hipStreamCreate(&streamEnc_));
  // hipStreamCreateWithFlags(stream_.get(), hipStreamNonBlocking);
}

CudaStreamHandler::~CudaStreamHandler()
{
  HANDLE_ERROR(hipStreamDestroy(stream_));
  HANDLE_ERROR(hipStreamDestroy(streamEnc_));
}

///////////////////////////////////////////////////////////////////////
CublasHandler::CublasHandler()
{
  hipblasStatus_t stat;
  stat = hipblasCreate(&handle_);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
  printf ("hipblasCreate initialization failed\n");
  abort();
  }

  stat = hipblasSetStream(handle_, CudaStreamHandler::GetStream());
  if (stat != HIPBLAS_STATUS_SUCCESS) {
  printf ("hipblasSetStream initialization failed\n");
  abort();
  }
}

CublasHandler::~CublasHandler() {
  hipblasDestroy(handle_);
}


}
}
}
